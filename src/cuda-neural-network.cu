#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <cuda-neural-network.h>

float *d_network, *d_training_dataset, *d_testing_dataset, *d_output, *_h_temp, *_d_temp;
int *_node_counts, *d_node_counts, *training_dataset_labels, training_dataset_size, *testing_dataset_labels, testing_dataset_size, _max_nodes, _network_size, _datum_size, _output_node_count;
activation *_activations, *d_activations;

#define BLOCK_SIZE 1024
#define LAYER_COUNT 4

__device__ float temp_array[BLOCK_SIZE];
__device__ float temp;

inline __device__ void deviceVectorDotProduct(int vector_size, float *a, float *b, int aInc, int bInc, float *c, int inRange){
    (*c) = 0.0;
    for(int i = 0; i < vector_size * inRange; i++){
        (*c) += a[aInc * i] * b[bInc * i];
    }
}

inline __device__ void deviceApplyPerceptron(float *a, int index, int nodeCount, activation perceptron){
    float *point = &a[index];
    if(perceptron == SoftMax){
        (*point) = pow(M_E, (*point));

        atomicAdd(&temp, *point);

        (*point) /= temp;
    }else{
        //ReLu branchless
        (*point) *= !((perceptron == ReLu) && ((*point) < 0.0));
        //Sigmoid branchless
        (*point) = (perceptron != Sigmoid) * (*point) + (perceptron == Sigmoid) * (*point);
    }
}

inline __device__ void deviceSwapPtrs(float **ptr1, float **ptr2){
    float *tmp = *ptr1;
    *ptr1 = *ptr2;
    *ptr2 = tmp;
}

inline __device__ void deviceEvalNeuralNetwork(float *input, float *network, int *nodeCounts, activation *perceptrons, float *output, int maxNodes){
    int x = threadIdx.x;
    int index = x;
    int lastNodeCount;
    int nodeCount = nodeCounts[0];
    int inRange = x < nodeCount;
    activation perceptron = perceptrons[0];

    if(inRange) deviceApplyPerceptron(output, x, nodeCount, perceptron);

    #pragma unroll
    for(int i = 1; i < LAYER_COUNT; i++){
        temp = 0.0;
        lastNodeCount = nodeCount;
        nodeCount = nodeCounts[i];
        inRange = x < nodeCount;
        perceptron = perceptrons[i];

        deviceVectorDotProduct(lastNodeCount, input, &network[index], 1, nodeCount, &output[x], inRange);

        __syncthreads();

        index += nodeCount * lastNodeCount;

        output[x] += inRange * network[index];

        __syncthreads();

        index += nodeCount;

        if(inRange) deviceApplyPerceptron(output, x, nodeCount, perceptron);

        __syncthreads();

        deviceSwapPtrs(&input, &output);
    }

    if(x < nodeCounts[LAYER_COUNT - 1] && LAYER_COUNT % 2 == 1) output[x] = input[x];
}

__global__ void cudaEvalNeuralNetwork(float *input, float *network, int *nodeCounts, activation *perceptrons, float *output, int maxNodes){
    int x = threadIdx.x;
    for(int i = 0; i < 100000; i++){
        temp_array[x] = input[x];
        __syncthreads();
        deviceEvalNeuralNetwork(temp_array, network, nodeCounts, perceptrons, output, maxNodes);
        __syncthreads();
    }
}

__global__ void cudaInc(float *a, float b){
    a[threadIdx.x] += b;
}

int mallocNetwork(int *nodeCounts, float **network){
    int i, j, k, weights;
    int networkSize = 0;

    #pragma unroll
    for(i = 1; i < LAYER_COUNT; i++){
        networkSize += nodeCounts[i] * (nodeCounts[i - 1] + 1);
    }

    *network = (float *)malloc(networkSize * sizeof(float));

    i = 0;
    #pragma unroll
    for(j = 1; j < LAYER_COUNT; j++){
        weights = nodeCounts[j - 1] * nodeCounts[j];
        
        for(k = 0; k < weights; k++){
            (*network)[i] = 1.0 / (float)nodeCounts[j - 1];
            i++;
        }

        for(k = 0; k < nodeCounts[j]; k++){
            (*network)[i] = 0.0;
            i++;
        }
    }

    return networkSize;
}

void initializeNetwork(float *h_network, activation *activations, int *node_counts){
    int i;
    _node_counts = (int*)malloc(LAYER_COUNT * sizeof(int));
    _activations = (activation*)malloc(LAYER_COUNT * sizeof(activation));
    
    #pragma unroll
    for(i = 0; i < LAYER_COUNT; i++){
        _node_counts[i] = node_counts[i];
        _activations[i] = activations[i];
    }

    hipMalloc(&d_node_counts, LAYER_COUNT * sizeof(int));
    hipMemcpy(d_node_counts, _node_counts, LAYER_COUNT * sizeof(int), hipMemcpyHostToDevice);
    hipMalloc(&d_activations, LAYER_COUNT * sizeof(activation));
    hipMemcpy(d_activations, _activations, LAYER_COUNT * sizeof(activation), hipMemcpyHostToDevice);
    
    _network_size = 0;
    _datum_size = _node_counts[0];
    _output_node_count = _node_counts[LAYER_COUNT - 1];
    _max_nodes = _node_counts[0];

    #pragma unroll
    for(i = 1; i < LAYER_COUNT; i++){
        _network_size += _node_counts[i] * (_node_counts[i - 1] + 1);
        if(_node_counts[i] > _max_nodes) _max_nodes = _node_counts[i];
    }

    hipMalloc(&d_network, _network_size * sizeof(float));
    hipMemcpy(d_network, h_network, _network_size * sizeof(float), hipMemcpyHostToDevice);

    _h_temp = (float*)malloc(_max_nodes * sizeof(float));

    hipMalloc(&_d_temp, _max_nodes * sizeof(float));
    hipMalloc(&d_output, _max_nodes * sizeof(float));
}

void getNetwork(float *h_network){
    hipMemcpy(h_network, d_network, _network_size * sizeof(float), hipMemcpyDeviceToHost);
}

void setTrainingData(float *h_training_dataset, int *_training_dataset_labels, int _training_dataset_size){
    training_dataset_size = _training_dataset_size;

    hipMalloc(&d_training_dataset, _datum_size * training_dataset_size * sizeof(float));
    hipMemcpy(d_training_dataset, h_training_dataset, _datum_size * training_dataset_size * sizeof(float), hipMemcpyHostToDevice);
    
    training_dataset_labels = (int*)malloc(training_dataset_size * sizeof(int));
    for(int i = 0; i < training_dataset_size; i++) training_dataset_labels[i] = _training_dataset_labels[i];
}

void freeTrainingData(){
    hipFree(d_training_dataset);
    free(training_dataset_labels);
}

void setTestingData(float *h_testing_dataset, int *_testing_dataset_labels, int _testing_dataset_size){
    testing_dataset_size = _testing_dataset_size;

    hipMalloc(&d_testing_dataset, _datum_size * testing_dataset_size * sizeof(float));
    hipMemcpy(d_testing_dataset, h_testing_dataset, _datum_size * testing_dataset_size * sizeof(float), hipMemcpyHostToDevice);
    
    testing_dataset_labels = (int*)malloc(testing_dataset_size * sizeof(int));
    for(int i = 0; i < testing_dataset_size; i++) testing_dataset_labels[i] = _testing_dataset_labels[i];
}

void freeTestingData(){
    hipFree(d_testing_dataset);
    free(testing_dataset_labels);
}

inline void _swapPointers(float **a, float **b){
    float *tmp = *a;
    *a = *b;
    *b = tmp;
}

inline void _evalCudaNeuralNetwork(float *d_input){
    cudaEvalNeuralNetwork<<<1, BLOCK_SIZE>>>(d_input, d_network, d_node_counts, d_activations, d_output, _max_nodes);
    hipDeviceSynchronize();
}

void printLastOut(){
    hipMemcpy(_h_temp, d_output, _output_node_count * sizeof(float), hipMemcpyDeviceToHost);
    for(int i = 0; i < _output_node_count; i++){
        printf("out[%d]: %f\n", i, _h_temp[i]);
    }
}

void evalCudaNeuralNetwork(float *d_input){
    _evalCudaNeuralNetwork(d_input);
}

inline float _lossCuda(float *d_dataset, int *dataset_labels, int datum_index){
    _evalCudaNeuralNetwork(&d_dataset[_datum_size * datum_index]);
    hipMemcpy(_h_temp, d_output, _output_node_count * sizeof(float), hipMemcpyDeviceToHost);

    return 1.0 - _h_temp[dataset_labels[datum_index]];
}

float lossCuda(float *d_dataset, int *dataset_labels, int datum_index){
    return _lossCuda(d_dataset, dataset_labels, datum_index);
}

float _batchLossCuda(float *d_dataset, int *dataset_labels, int *batch, int batchSize){
    float loss = 0.0;

    for(int i = 0; i < batchSize; i++){
        loss += _lossCuda(d_dataset, dataset_labels, batch[i]);
    }

    return loss / (float)batchSize;
}

void trainCuda(int *batch, int batchSize, float trainingSpeed){
    int point;
    float deltaLoss, gradient;
    float currentLoss = _batchLossCuda(d_training_dataset, training_dataset_labels, batch, batchSize);

    for(int i = 0; i < STOCHASTIC_POINTS; i++){
        point = rand() % _network_size;
        for(int j = 0; j < STOCHASTIC_STEPS; j++){
            cudaInc<<<1, 1>>>(&d_network[point], DELTA_WEIGHT);
            deltaLoss = _batchLossCuda(d_training_dataset, training_dataset_labels, batch, batchSize) - currentLoss;
            gradient = deltaLoss / DELTA_WEIGHT;
            if(fabs(gradient) < 0.001) break;
            cudaInc<<<1, 1>>>(&d_network[point], -(gradient * trainingSpeed + DELTA_WEIGHT));
        }
    }
}

float accuracy(){
    int correct = 0;
    float v;
    int j, prediction;
    float *h_output = (float*)malloc(_output_node_count * sizeof(float));

    for(int i = 0; i < testing_dataset_size; i++){
        _evalCudaNeuralNetwork(&d_testing_dataset[_datum_size * i]);
        hipMemcpy(h_output, d_output, _output_node_count * sizeof(float), hipMemcpyDeviceToHost);
        
        v = h_output[0];
        prediction = 0;

        for(j = 1; j < _node_counts[LAYER_COUNT - 1]; j++){
            if(h_output[j] > v){
                v = h_output[j];
                prediction = j;
            }
        }

        if(prediction == testing_dataset_labels[i]) correct++;
    }

    return (float)correct / (float)testing_dataset_size;
}

float confidenceRating(float *output, int *prediction){
    (*prediction) = 0;
    float total = 0.0;

    for(int i = 0; i < _output_node_count; i++){
        total += output[i];
        if(output[i] > output[*prediction]) (*prediction) = i;
    }

    float lowerBound = total / (float)_output_node_count;

    return (output[*prediction] - lowerBound) / (total - lowerBound);
}